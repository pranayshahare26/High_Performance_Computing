
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define ARRSIZE 20
#define BLOCK_SIZE 5

__global__ void vectorAdd(int *A, int *B, int *C, int elements_per_thread)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * elements_per_thread;
    int end = start + elements_per_thread;
     for (int i = start; i < end; i++)
    {
        C[i] = A[i] + B[i];
    }
}
int main()
{
    int *A, *B, *C;
    int elements_per_thread = ARRSIZE / BLOCK_SIZE;
    A = (int *)malloc(ARRSIZE * sizeof(int));
    B = (int *)malloc(ARRSIZE * sizeof(int));
    C = (int *)malloc(ARRSIZE * sizeof(int));
     // Initialize data to some value
    for (int i = 0; i < ARRSIZE; i++)
    {
        A[i] = i;
        B[i] = i;
    }
    printf("Initial data: \n");
    for (int i = 0; i < ARRSIZE; i++)
    {
        printf("%d ", A[i]);
    }
    printf("\n");
    for (int i = 0; i < ARRSIZE; i++)
    {
        printf("%d ", B[i]);
    }
    printf("\n");
    int *d_A, *d_B, *d_C;
    double exe_time;
    struct timeval start_time, stop_time;

    hipMalloc((void **)&d_A, ARRSIZE * sizeof(int));
    hipMalloc((void **)&d_B, ARRSIZE * sizeof(int));
    hipMalloc((void **)&d_C, ARRSIZE * sizeof(int));
    hipMemcpy(d_A, A, ARRSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, ARRSIZE * sizeof(int), hipMemcpyHostToDevice);

    gettimeofday(&start_time, NULL);
    vectorAdd<<<BLOCK_SIZE, elements_per_thread>>>(d_A, d_B, d_C, elements_per_thread);
    gettimeofday(&stop_time, NULL);
    exe_time = (stop_time.tv_sec + (stop_time.tv_usec / 1000000.0)) - (start_time.tv_sec + (start_time.tv_usec / 1000000.0));

    printf("Vector addition output: \n");
    hipMemcpy(C, d_C, ARRSIZE * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < ARRSIZE; i++)
    {
        printf("%d ", C[i]);
    }
    printf("\n");
    printf("Execution time is = %lf seconds\n", exe_time);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    return 0;
}

/*

The following code performs vector addition using CUDA. It initializes two arrays, A and B, with values from 0 to 19. It then allocates memory on the GPU for arrays A, B, and C, and copies the data from the CPU to the GPU. The vectorAdd kernel function is then called, which adds the corresponding elements of A and B and stores the result in C. The execution time of the kernel is measured using gettimeofday. Finally, the result is copied back from the GPU to the CPU and printed along with the execution time.
 Step-wise explanation of the code:
1. Include necessary header files: stdio.h, stdlib.h, sys/time.h.
2. Define constants ARRSIZE and BLOCK_SIZE.
3. Define the vectorAdd kernel function, which takes three integer pointers A, B, and C, and an integer elements_per_thread.
4. Inside the kernel function, calculate the thread index (tid) using threadIdx.x and blockIdx.x, and the start and end indices for the thread's portion of the arrays.
5. Use a for loop to add the corresponding elements of A and B and store the result in C.
6. Define the main function.
7. Declare pointers A, B, and C for the arrays on the CPU.
8. Calculate the number of elements per thread by dividing ARRSIZE by BLOCK_SIZE.
9. Allocate memory on the CPU for arrays A, B, and C using malloc.
10. Initialize the data in arrays A and B with values from 0 to 19.
11. Print the initial data in arrays A and B.
12. Declare pointers d_A, d_B, and d_C for the arrays on the GPU.
13. Declare variables exe_time, start_time, and stop_time for measuring the execution time.
14. Allocate memory on the GPU for arrays d_A, d_B, and d_C using cudaMalloc.
15. Copy the data from arrays A and B on the CPU to arrays d_A and d_B on the GPU using cudaMemcpy.
16. Measure the start time using gettimeofday.
17. Call the vectorAdd kernel function with BLOCK_SIZE blocks and elements_per_thread threads per block.
18. Measure the stop time using gettimeofday.
19. Calculate the execution time by subtracting the start time from the stop time.
20. Print the vector addition output from array C on the CPU.
21. Copy the result from array d_C on the GPU to array C on the CPU using cudaMemcpy.
22. Print the execution time.
23. Free the memory on the GPU using cudaFree.
24. Free the memory on the CPU using free.
25. Return 0 to indicate successful execution.

*/