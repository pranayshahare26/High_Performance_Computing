
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define ARRSIZE 100000

__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid = threadIdx.x;
	pd[myid] = md[myid] + nd[myid];
}


int test_result(int *m, int *n, int *p, int *md, int *nd, int *pd, int num_threads)
{
	int size = num_threads*sizeof(int);
	int i, flag;
	flag = 0;
	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(num_threads, 1);

	arradd<<< DimGrid,DimBlock >>>(md,nd,pd);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	
	for(i=0;i<num_threads;i++)
	{
		if(p[i] != 2*i)
		{
			flag = 1;
			break;
		}
	}
	return flag;
}
int main()
{
	//int size = ARRSIZE * sizeof(int);
	int m[ARRSIZE], n[ARRSIZE], p[ARRSIZE],*md, *nd,*pd;
	int i=0;
    int start, end, num_threads;
    int flag;
	for(i=0; i<ARRSIZE; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}
    start = 1;
    end = 10000;
    printf("\n Choosing Start...");
    while(1)
    {   
        printf("\n Start = %d", start);
        flag = test_result(m, n, p, md, nd, pd, start);
	    if(flag != 0)   // Answers match
	    {
	        start = start/2;
	    }
	    else
	    {
	        break;
	    }
    }    
    printf("\n Choosing End...");
    while(1)
    {
	    printf("\n End = %d", end);
        flag = test_result(m, n, p, md, nd, pd, end);
	    if(flag == 0)   // Answers match
	    {
	        end = end *2;
	    }
	    else
	    {
	        break;
	    }	    
    }
    printf("\n Setting Start = %d and End = %d", start, end);
    while(1)
    {
	    num_threads = (start + end)/2;
	    printf("\n Start = %d, End = %d, num_threads = %d", start, end,num_threads);
	    flag = test_result(m, n, p, md, nd, pd, num_threads);
	    if(flag == 0)   // Answers match
	    {
	        start = num_threads;
	        if((start == end) || ((end-start) == 1))
	        {
	            break;
	        }
	    }
	    else
	    {
	        end = num_threads;
	    }
    }
	printf("\nFinal value of num_threads = %d \n", num_threads);
		
	hipFree(md); 
	hipFree(nd);
	hipFree(pd);	
}

/*

The code is performing a binary search to find the optimal number of threads for the kernel function "arradd". 
The "arradd" function is a CUDA kernel function that takes three integer arrays as input: "md", "nd", and "pd".
It calculates the sum of the corresponding elements of "md" and "nd" and stores the result in "pd".
The "test_result" function is used to test the correctness of the "arradd" function.
It takes six integer pointers as input: "m", "n", "p", "md", "nd", and "pd", and the number of threads.
It allocates memory on the GPU for "md", "nd", and "pd", copies the data from the host to the GPU, calls the "arradd" kernel function, copies the result back to the host, and checks if the result is correct.
If the result is not correct, it sets the "flag" variable to 1. Finally, it returns the value of "flag".
In the "main" function, the arrays "m", "n", and "p" are initialized with values from 0 to ARRSIZE-1. 
The variables "start" and "end" are initialized with values 1 and 10000 respectively.
The code then performs a binary search to find the optimal number of threads for the "arradd" kernel function. 
In the first while loop, the code repeatedly calls the "test_result" function with different values of "start" and checks if the result is correct. 
If the result is correct, it divides the value of "start" by 2. This loop continues until the result is not correct.
In the second while loop, the code repeatedly calls the "test_result" function with different values of "end" and checks if the result is correct. 
If the result is correct, it multiplies the value of "end" by 2. This loop continues until the result is not correct.
In the third while loop, the code calculates the value of "num_threads" as the average of "start" and "end". 
It calls the "test_result" function with this value and checks if the result is correct.
If the result is correct, it updates the value of "start" to "num_threads". If the difference between "end" and "start" is 1 or "start" is equal to "end", the loop breaks.
Otherwise, it updates the value of "end" to "num_threads".
Finally, the code prints the final value of "num_threads" and frees the allocated memory on the GPU.
Overall, the code is trying to find the optimal number of threads for the "arradd" kernel function by performing a binary search.

*/