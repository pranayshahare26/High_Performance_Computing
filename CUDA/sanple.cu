#include <hip/hip_runtime.h>
#include <stdio.h>
// __global__ void matrixAddition(float* A, const float* B, const float* C, int size) 
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     if (idx < size && idy < size) 
//     {
//         int index = idy * size + idx;
//         A[index] = B[index] + C[index];
//     }
// }
// // Kernel-Each thread produces one output matrix element
// __global__ void matrixAddition(float* A, const float* B, const float* C, int size) 
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     if (idx < size && idy < size) 
//     {
//         int index = idy * size + idx;
//         A[index] = B[index] + C[index];
//     }
// }

// // Kernal-Each thread produces one output matrix row
// __global__ void matrixAddition(float* A, const float* B, const float* C, int size) 
// {
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     if (idy < size) 
//     {
//         for (int idx = 0; idx < size; idx++) 
//         {
//             int index = idy * size + idx;
//             A[index] = B[index] + C[index];
//         }
//     }
// }

// Kernel-Each thread produces one output matrix column
__global__ void matrixAddition(float* A, const float* B, const float* C, int size) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) 
    {
        for (int idy = 0; idy < size; idy++) 
        {
            int index = idy * size + idx;
            A[index] = B[index] + C[index];
        }
    }
}

void matrixAdditionHost(float* A, const float* B, const float* C, int size) 
{
    float* d_A, *d_B, *d_C;
    int matrixSize = size * size * sizeof(float);
    hipMalloc((void**)&d_A, matrixSize);
    hipMalloc((void**)&d_B, matrixSize);
    hipMalloc((void**)&d_C, matrixSize);
    hipMemcpy(d_B, B, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, matrixSize, hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x, (size + blockSize.y - 1) / blockSize.y);
    matrixAddition<<<gridSize, blockSize>>>(d_A, d_B, d_C, size);
    hipMemcpy(A, d_A, matrixSize, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
int main() 
{
    int size = 4;
    float* A = new float[size * size];
    float* B = new float[size * size];
    float* C = new float[size * size];
     // Initialize B and C matrices
     matrixAdditionHost(A, B, C, size);
     // Print the resulting matrix A
    delete[] A;
    delete[] B;
    delete[] C;
    return 0;
}