#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N 1000000
#define THREADS_PER_BLOCK 256

__global__ void calculate_pi(double step, double* sum) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double x = (i + 0.5) * step;
    if (i < N) 
    {
        sum[i] = 4.0 / (1.0 + x * x);
    }
}
int main() 
{
    double step = 1.0 / (double)N;
    double* sum_host = (double*)malloc(N * sizeof(double));
    double* sum_device;
    hipMalloc(&sum_device, N * sizeof(double));
    int num_blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    calculate_pi<<<num_blocks, THREADS_PER_BLOCK>>>(step, sum_device);
    hipMemcpy(sum_host, sum_device, N * sizeof(double), hipMemcpyDeviceToHost);
    double pi = 0.0;
    for (int i = 0; i < N; i++) 
    {
        pi += sum_host[i];
    }
    pi *= step;
    printf("Approximate Value of pi is %f.\n", pi);
    
    free(sum_host);
    hipFree(sum_device);
    return 0;
}