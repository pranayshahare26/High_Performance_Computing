
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define N 1000000
#define THREADS_PER_BLOCK 256
__global__ void calculate_pi(double step, double* sum) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double x = (i + 0.5) * step;
    if (i < N) 
    {
        sum[i] = 4.0 / (1.0 + x * x);
    }
}
double get_time() 
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec / 1000000.0;
}
int main() 
{
    double step = 1.0 / (double)N;
    double* sum_host = (double*)malloc(N * sizeof(double));
    double* sum_device;
    
    hipMalloc(&sum_device, N * sizeof(double));
    int num_blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    double start_time = get_time();
    calculate_pi<<<num_blocks, THREADS_PER_BLOCK>>>(step, sum_device);
    
    hipMemcpy(sum_host, sum_device, N * sizeof(double), hipMemcpyDeviceToHost);
    
    double pi = 0.0;
    for (int i = 0; i < N; i++) 
    {
        pi += sum_host[i];
    }
    pi *= step;
    
    double end_time = get_time();
    double exe_time = end_time - start_time;
    
    printf("Approximate Value of pi is %f.\n", pi);
    printf("Execution time: %lf seconds\n", exe_time);
    
    free(sum_host);
    hipFree(sum_device);
    return 0;
}

/*

This code calculates an approximate value of pi using the Monte Carlo method in parallel using CUDA.
Here is a step-by-step explanation of the code:
1. The code includes necessary header files and defines constants N (number of iterations) and THREADS_PER_BLOCK (number of threads per block in the GPU).
2. The code defines a CUDA kernel function  `calculate_pi`  which takes in the step size and a pointer to the device memory for storing the intermediate sums. The function calculates the value of pi for each thread using the Monte Carlo method and stores it in the device memory.
3. The code defines a helper function  `get_time`  which returns the current time in seconds.
4. In the  `main`  function:
   - The step size is calculated as 1.0 divided by the number of iterations.
   - Memory is allocated on the host for storing the intermediate sums.
   - Memory is allocated on the device for storing the intermediate sums.
   - The number of blocks required for the given number of iterations is calculated.
   - The start time is recorded using  `get_time` .
   - The  `calculate_pi`  kernel is launched on the device with the specified number of blocks and threads per block, passing the step size and device memory pointer.
   - The intermediate sums are copied from the device memory to the host memory.
   - The final value of pi is calculated by summing up all the intermediate sums and multiplying by the step size.
   - The end time is recorded using  `get_time` .
   - The execution time is calculated as the difference between the end time and start time.
   - The approximate value of pi and the execution time are printed.
   - The host and device memory are freed.
In summary, this code uses CUDA to calculate an approximate value of pi in parallel by dividing the work among multiple threads on the GPU. The intermediate sums are calculated in parallel and then combined on the host to obtain the final result. The execution time is also measured and printed.

*/