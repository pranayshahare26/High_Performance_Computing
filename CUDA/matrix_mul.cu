
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define VECTORSIZE 10

__global__ void matrix_vec_mul(int *A, int *B, int *C)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	int sum = 0;
	if(i<VECTORSIZE)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		C[i] =  sum;
	}
}

int main(int argc, char **argv)
{
	int i, j;
	int *A, *B, *C, *Ad, *Bd, *Cd; 	
	double exe_time;
	struct timeval stop_time, start_time;
	
	//Allocate and initialize the arrays
	A = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	B = (int *)malloc(VECTORSIZE*sizeof(int));
	C = (int *)malloc(VECTORSIZE*sizeof(int));
	
	//Initialize data to some value
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			A[i*VECTORSIZE+j] = 1;	
		}
		B[i] = 1;
	}
	
	//print the data
	/*printf("\nInitial data: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", A[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}
	printf("\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", B[i]);
	}*/
	
	gettimeofday(&start_time, NULL);
	
	/*for(i=0;i<VECTORSIZE;i++)
	{
		sum = 0;
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		C[i] =  sum;
	}*/
	
	hipMalloc(&Ad,VECTORSIZE*VECTORSIZE*sizeof(int));
	hipMalloc(&Bd,VECTORSIZE*sizeof(int));
	hipMalloc(&Cd,VECTORSIZE*sizeof(int));
	
	hipMemcpy(Ad,A,VECTORSIZE*VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
	
	int Total_num_Threads = VECTORSIZE;
	int num_threads_per_block = 256;
	int numblocks = Total_num_Threads/num_threads_per_block + 1;
	
	matrix_vec_mul<<<numblocks,num_threads_per_block>>>(Ad,Bd,Cd);
	
	hipMemcpy(C,Cd,VECTORSIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	//print the data
	/*printf("\n Matrix-Vector Multiplication output: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", C[i]);	
	}*/
	printf("\nExecution time is = %lf seconds\n", exe_time);	
	printf("\nProgram exit!\n");
	
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
	
	//Free arrays
	free(A); 
	free(B);
	free(C);
}

/*

The code is performing matrix-vector multiplication using CUDA. 
 1. The code defines the size of the vectors as VECTORSIZE.
2. The code then defines a CUDA kernel function called matrix_vec_mul, which takes three integer pointers as input: A, B, and C.
3. Inside the kernel function, it calculates the index i based on the block and thread indices, and initializes the sum variable.
4. It checks if i is less than VECTORSIZE, and if so, it enters a loop to calculate the sum by multiplying each element of row i in matrix A with the corresponding element in vector B, and adding it to the sum.
5. Finally, it stores the sum in the corresponding index of vector C.
6. The main function starts by declaring variables and allocating memory for arrays A, B, and C.
7. It then initializes the data in arrays A and B with the value 1.
8. It allocates memory on the GPU for arrays Ad, Bd, and Cd, and copies the data from arrays A and B on the host to the corresponding arrays on the GPU.
9. It calculates the number of blocks and threads per block needed to process the data, and calls the matrix_vec_mul kernel function with the specified number of blocks and threads per block.
10. It then copies the result from array Cd on the GPU back to array C on the host.
11. It calculates the execution time by measuring the start and stop time using the gettimeofday function.
12. It prints the execution time.
13. It frees the memory on the GPU.
14. It frees the memory for arrays A, B, and C on the host.

*/