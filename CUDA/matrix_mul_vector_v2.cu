
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define VECTORSIZE 10000
#define NUM_THDS_PER_BLK 256

__global__ void matrix_vec_mul(int *A, int *B, int *C)
{
	int i = blockIdx.x;
	int j;
	int sum = 0;
	int range = NUM_THDS_PER_BLK/2;
	__shared__ int sum_arr[NUM_THDS_PER_BLK];
	sum_arr[threadIdx.x] = 0;
	if(blockIdx.x < VECTORSIZE)
	{
		for(j=threadIdx.x;j<VECTORSIZE;j+=blockDim.x)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		sum_arr[threadIdx.x] = sum;
		__syncthreads();

		while(range>0)
        {
            if(threadIdx.x < range)
            {
                sum_arr[threadIdx.x] += sum_arr[threadIdx.x + range];
            }
            range = range /2;
            __syncthreads(); 
        }
		
		if(threadIdx.x == 0)
		{
		    C[i] =  sum_arr[0];
	    }
	}
}
int main(int argc, char **argv)
{
	int i, j;
	int *A, *B, *C, *Ad, *Bd, *Cd; 	
	double exe_time;
	struct timeval stop_time, start_time;
	
	//Allocate and initialize the arrays
	A = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	B = (int *)malloc(VECTORSIZE*sizeof(int));
	C = (int *)malloc(VECTORSIZE*sizeof(int));
	
	//Initialize data to some value
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			A[i*VECTORSIZE+j] = 1;	
		}
		B[i] = 1;
	}
	
	//print the data
	/*printf("\nInitial data: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", A[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}
	printf("\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", B[i]);
	}*/	
	
	gettimeofday(&start_time, NULL);
	
	/*for(i=0;i<VECTORSIZE;i++)
	{
		sum = 0;
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = sum + A[i*VECTORSIZE+j]*B[i];	
		}
		C[i] =  sum;
	}*/
	
	hipMalloc(&Ad,VECTORSIZE*VECTORSIZE*sizeof(int));
	hipMalloc(&Bd,VECTORSIZE*sizeof(int));
	hipMalloc(&Cd,VECTORSIZE*sizeof(int));
	
	hipMemcpy(Ad,A,VECTORSIZE*VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,VECTORSIZE*sizeof(int),hipMemcpyHostToDevice);
	
	int num_threads_per_block = NUM_THDS_PER_BLK;
	int numblocks = VECTORSIZE;
	
	matrix_vec_mul<<<numblocks,num_threads_per_block>>>(Ad,Bd,Cd);
	
	hipMemcpy(C,Cd,VECTORSIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	//print the data
	/*printf("\n Matrix-Vector Multiplication output: \n");
	for(i=0;i<VECTORSIZE;i++)
	{
		printf("\t%d", C[i]);	
	}*/
	printf("\nExecution time is = %lf seconds\n", exe_time);
	printf("\nProgram exit!\n");
	
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
	
	//Free arrays
	free(A); 
	free(B);
	free(C);
}

/*

The code is a CUDA program that performs matrix-vector multiplication. 
It multiplies a matrix A of size VECTORSIZE x VECTORSIZE with a vector B of size VECTORSIZE to produce a vector C of size VECTORSIZE.
The main function starts by allocating and initializing the arrays A, B, and C.
A is initialized as a matrix of all 1s, and B is initialized as a vector of all 1s.
Then, the code measures the execution time using gettimeofday function.
Next, it allocates memory on the GPU for the arrays Ad, Bd, and Cd using cudaMalloc function.
It then copies the data from the CPU arrays A and B to the GPU arrays Ad and Bd using cudaMemcpy function.
The code defines the number of total threads as VECTORSIZE and the number of threads per block as NUM_THDS_PER_BLK.
It calculates the number of blocks needed as VECTORSIZE.
Then, it calls the kernel function matrix_vec_mul with the specified number of blocks and threads per block.
The kernel function performs the matrix-vector multiplication using parallel threads on the GPU.
Each thread calculates a partial sum and stores it in the shared array sum_arr.
The shared array is then reduced using a binary tree reduction algorithm until only one thread remains. 
The result is then stored in the output array C.
After the kernel execution, the result is copied back from the GPU array Cd to the CPU array C using cudaMemcpy function.
Finally, the code measures the execution time again and prints it along with a message indicating the program exit.
It also frees the allocated memory on the GPU using cudaFree function, and frees the arrays A, B, and C on the CPU using free function.

*/