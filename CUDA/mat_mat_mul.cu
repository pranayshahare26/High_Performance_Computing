
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define VECTORSIZE 	5000
#define NUM_THDS	256

__global__ void matmul(int *A, int *B, int *C)
{
	int myid = blockIdx.x*blockDim.x+threadIdx.x;
	int i = myid / VECTORSIZE;	
	int j = myid % VECTORSIZE;
	int k;
	int sum = 0;
	if(myid < (VECTORSIZE*VECTORSIZE))
	{
		for(k=0;k<VECTORSIZE;k++)
		{
			sum = sum + A[i*VECTORSIZE+k]*B[k*VECTORSIZE+j];	
		}
		C[i*VECTORSIZE+j] =  sum;
	}
}

int main(int argc, char **argv)
{
	int i, j;
	int *A, *B, *C, *Ad, *Bd, *Cd;		
	double exe_time;
	struct timeval stop_time, start_time;
	
	//Allocate and initialize the arrays
	A = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	B = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	C = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	
	//Initialize data to some value
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			A[i*VECTORSIZE+j] = 1;
			B[i*VECTORSIZE+j] = 2;
			C[i*VECTORSIZE+j] = 0;	
		}
	}
	
	//print the data
	/*printf("\nInitial data: \n");
	printf("\n A matrix:\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", A[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}
	printf("\n B matrix:\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", B[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}*/	
	
	gettimeofday(&start_time, NULL);
	
	int total_threads = VECTORSIZE*VECTORSIZE;
	int num_thds_per_block = NUM_THDS; 
	int num_blocks = total_threads / num_thds_per_block + 1;
	
	hipMalloc(&Ad, VECTORSIZE*VECTORSIZE*sizeof(int));
	hipMemcpy(Ad, A, VECTORSIZE*VECTORSIZE*sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc(&Bd, VECTORSIZE*VECTORSIZE*sizeof(int));
	hipMemcpy(Bd, B, VECTORSIZE*VECTORSIZE*sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc(&Cd, VECTORSIZE*VECTORSIZE*sizeof(int));
	
	matmul<<<num_blocks,num_thds_per_block>>>(Ad,Bd,Cd);
	
	hipMemcpy(C, Cd, VECTORSIZE*VECTORSIZE*sizeof(int), hipMemcpyDeviceToHost);
		
	/*for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			sum = 0;
			for(k=0;k<VECTORSIZE;k++)
			{
				sum = sum + A[i*VECTORSIZE+k]*B[k*VECTORSIZE+j];	
			}
			C[i*VECTORSIZE+j] =  sum;
		}
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	//print the data
	/*printf("\n C matrix:\n");
	for(i=0;i<VECTORSIZE;i++)
	{
		for(j=0;j<VECTORSIZE;j++)
		{
			printf("\t%d ", C[i*VECTORSIZE+j]);	
		}
		printf("\n");
	}*/
	
	printf("\nC[5*VECTORSIZE+5] = %d ", C[5*VECTORSIZE+5]);	
	
	printf("\nExecution time is = %lf seconds\n", exe_time);
	
	printf("\nProgram exit!\n");
	
	//Free arrays
	free(A); 
	free(B);
	free(C);
	hipFree(Ad); 
	hipFree(Bd);
	hipFree(Cd);
}

/*

The code is performing matrix multiplication using CUDA parallel programming. It defines the size of the vectors and the number of threads to be used. It then defines a kernel function  `matmul`  which performs the matrix multiplication for a given block and thread. The main function initializes the arrays A, B, and C, and then allocates memory on the GPU for A, B, and C. It copies the data from the host to the device, calls the  `matmul`  kernel function, and then copies the result back to the host. Finally, it calculates the execution time and prints the result.
Step-wise explanation of the code:
1. Include necessary header files:  `stdio.h` ,  `stdlib.h` ,  `sys/time.h` .
2. Define the size of the vectors ( `VECTORSIZE` ) and the number of threads ( `NUM_THDS` ).
3. Define the kernel function  `matmul`  which performs matrix multiplication for a given block and thread.
4. Define the main function.
5. Declare variables:  `i` ,  `j` ,  `A` ,  `B` ,  `C` ,  `Ad` ,  `Bd` ,  `Cd` ,  `exe_time` ,  `stop_time` ,  `start_time` .
6. Allocate memory for arrays A, B, and C using  `malloc` .
7. Initialize the data in arrays A, B, and C.
8. Get the start time using  `gettimeofday` .
9. Calculate the total number of threads, number of threads per block, and number of blocks.
10. Allocate memory on the GPU for A, B, and C using  `cudaMalloc` .
11. Copy data from host to device using  `cudaMemcpy` .
12. Call the  `matmul`  kernel function with the specified number of blocks and threads.
13. Copy the result from the device to the host using  `cudaMemcpy` .
14. Get the stop time using  `gettimeofday` .
15. Calculate the execution time.
16. Print the result of C[5*VECTORSIZE+5].
17. Print the execution time.
18. Free the allocated memory for A, B, and C using  `free` .
19. Free the allocated memory on the GPU using  `cudaFree` .
20. End the program.

*/