
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define ARRSIZE 20
#define BLOCK_SIZE 5

__global__ void vectorAdd(int *A, int *B, int *C, int elements_per_thread)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * elements_per_thread;
    int end = start + elements_per_thread;
     for (int i = start; i < end; i++)
    {
        C[i] = A[i] + B[i];
    }
}
int main()
{
    int *A, *B, *C;
    int elements_per_thread = ARRSIZE / BLOCK_SIZE;
    A = (int *)malloc(ARRSIZE * sizeof(int));
    B = (int *)malloc(ARRSIZE * sizeof(int));
    C = (int *)malloc(ARRSIZE * sizeof(int));
     // Initialize data to some value
    for (int i = 0; i < ARRSIZE; i++)
    {
        A[i] = i;
        B[i] = i;
    }
    printf("Initial data: \n");
    for (int i = 0; i < ARRSIZE; i++)
    {
        printf("%d ", A[i]);
    }
    printf("\n");
    for (int i = 0; i < ARRSIZE; i++)
    {
        printf("%d ", B[i]);
    }
    printf("\n");
    int *d_A, *d_B, *d_C;
    double exe_time;
    struct timeval start_time, stop_time;

    hipMalloc((void **)&d_A, ARRSIZE * sizeof(int));
    hipMalloc((void **)&d_B, ARRSIZE * sizeof(int));
    hipMalloc((void **)&d_C, ARRSIZE * sizeof(int));
    hipMemcpy(d_A, A, ARRSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, ARRSIZE * sizeof(int), hipMemcpyHostToDevice);

    gettimeofday(&start_time, NULL);
    vectorAdd<<<BLOCK_SIZE, elements_per_thread>>>(d_A, d_B, d_C, elements_per_thread);
    gettimeofday(&stop_time, NULL);
    exe_time = (stop_time.tv_sec + (stop_time.tv_usec / 1000000.0)) - (start_time.tv_sec + (start_time.tv_usec / 1000000.0));

    printf("Vector addition output: \n");
    hipMemcpy(C, d_C, ARRSIZE * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < ARRSIZE; i++)
    {
        printf("%d ", C[i]);
    }
    printf("\n");
    printf("Execution time is = %lf seconds\n", exe_time);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    return 0;
}
