
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define VECTORSIZE 10000
void matrix_vec_mul(int *A, int *B, int *C)
{
    int i, j, sum;
    for(i=0;i<VECTORSIZE;i++)
    {
        sum = 0;
        for(j=0;j<VECTORSIZE;j++)
        {
            sum = sum + A[i*VECTORSIZE+j]*B[j];    
        }
        C[i] =  sum;
    }
}
int main(int argc, char **argv)
{
    int i, j;
    int *A, *B, *C;     
    double exe_time;
    struct timeval stop_time, start_time;
     //Allocate and initialize the arrays
    A = (int *)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
    B = (int *)malloc(VECTORSIZE*sizeof(int));
    C = (int *)malloc(VECTORSIZE*sizeof(int));
     //Initialize data to some value
    for(i=0;i<VECTORSIZE;i++)
    {
        for(j=0;j<VECTORSIZE;j++)
        {
            A[i*VECTORSIZE+j] = 1;    
        }
        B[i] = 1;
    }
     //print the data
    /*printf("\nInitial data: \n");
    for(i=0;i<VECTORSIZE;i++)
    {
        for(j=0;j<VECTORSIZE;j++)
        {
            printf("\t%d ", A[i*VECTORSIZE+j]);    
        }
        printf("\n");
    }
    printf("\n");
    for(i=0;i<VECTORSIZE;i++)
    {
        printf("\t%d", B[i]);
    }*/    
    gettimeofday(&start_time, NULL);
    matrix_vec_mul(A, B, C);
    gettimeofday(&stop_time, NULL);    
    exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
     //print the data
    /*printf("\n Matrix-Vector Multiplication output: \n");
    for(i=0;i<VECTORSIZE;i++)
    {
        printf("\t%d", C[i]);    
    }*/
    printf("\nExecution time is = %lf seconds\n", exe_time);
    printf("\nProgram exit!\n");
     //Free arrays
    free(A); 
    free(B);
    free(C);
}

/*

The following code performs matrix-vector multiplication. It defines a function  `matrix_vec_mul`  that takes three integer arrays  `A` ,  `B` , and  `C`  as input. The function multiplies matrix  `A`  with vector  `B`  and stores the result in vector  `C` . 
 The main function starts by declaring variables and allocating memory for arrays  `A` ,  `B` , and  `C` . It then initializes the data in arrays  `A`  and  `B`  with the value 1. 
 Next, the code measures the execution time of the  `matrix_vec_mul`  function using the  `gettimeofday`  function. It calls the  `matrix_vec_mul`  function and passes the arrays  `A` ,  `B` , and  `C`  as arguments. After the function completes, it measures the stop time and calculates the execution time. 
 Finally, the code prints the execution time and frees the allocated memory for arrays  `A` ,  `B` , and  `C` .
 Step-wise explanation of the code:
1. The code includes necessary header files:  `stdio.h` ,  `stdlib.h` , and  `sys/time.h` .
2. It defines a constant  `VECTORSIZE`  as 10000.
3. It defines the function  `matrix_vec_mul`  that takes three integer arrays  `A` ,  `B` , and  `C`  as input.
4. Inside the  `matrix_vec_mul`  function, it initializes variables  `i` ,  `j` , and  `sum` .
5. It uses nested loops to perform matrix-vector multiplication. The outer loop iterates over each row of matrix  `A` , and the inner loop iterates over each element of vector  `B` .
6. Inside the inner loop, it multiplies each element of matrix  `A`  with the corresponding element of vector  `B`  and adds it to the  `sum`  variable.
7. After the inner loop completes, it assigns the value of  `sum`  to the corresponding element of vector  `C` .
8. The main function starts by declaring variables  `i` ,  `j` , and  `sum` .
9. It declares three integer pointers  `A` ,  `B` , and  `C` .
10. It declares variables  `exe_time` ,  `start_time` , and  `stop_time`  of types  `double`  and  `struct timeval` .
11. It allocates memory for arrays  `A` ,  `B` , and  `C`  using the  `malloc`  function.
12. It initializes the data in arrays  `A`  and  `B`  with the value 1 using nested loops.
13. It measures the start time using the  `gettimeofday`  function.
14. It calls the  `matrix_vec_mul`  function and passes arrays  `A` ,  `B` , and  `C`  as arguments.
15. It measures the stop time using the  `gettimeofday`  function.
16. It calculates the execution time by subtracting the start time from the stop time and stores it in the  `exe_time`  variable.
17. It prints the execution time.
18. It frees the allocated memory for arrays  `A` ,  `B` , and  `C`  using the  `free`  function.

*/