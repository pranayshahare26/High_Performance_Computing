
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define VECTORSIZE 10000

 __global__ void matrix_multiply(int *A, int *B, int *C, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j, k, sum;
    if (i < size)
    {
        for (j = 0; j < size; j++)
        {
            sum = 0;
            for (k = 0; k < size; k++)
            {
                sum += A[i * size + k] * B[k * size + j];
            }
            C[i * size + j] = sum;
        }
    }
}
 int main(int argc, char **argv)
{
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    double exe_time;
    struct timeval stop_time, start_time;
     //Allocate and initialize the arrays
    A = (int *)malloc(VECTORSIZE * VECTORSIZE * sizeof(int));
    B = (int *)malloc(VECTORSIZE * VECTORSIZE * sizeof(int));
    C = (int *)malloc(VECTORSIZE * VECTORSIZE * sizeof(int));
     //Initialize data to some value
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            A[i * VECTORSIZE + j] = 1;
            B[i * VECTORSIZE + j] = 2;
            C[i * VECTORSIZE + j] = 0;
        }
    }
     //print the data
  /*  printf("\nInitial data: \n");
    printf("\n A matrix:\n");
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            printf("\t%d ", A[i * VECTORSIZE + j]);
        }
        printf("\n");
    }
    printf("\n B matrix:\n");
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            printf("\t%d ", B[i * VECTORSIZE + j]);
        }
        printf("\n");
    }*/
     //Allocate memory on the device
    hipMalloc((void **)&d_A, VECTORSIZE * VECTORSIZE * sizeof(int));
    hipMalloc((void **)&d_B, VECTORSIZE * VECTORSIZE * sizeof(int));
    hipMalloc((void **)&d_C, VECTORSIZE * VECTORSIZE * sizeof(int));

     // Copy data from host to device
    hipMemcpy(d_A, A, VECTORSIZE * VECTORSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, VECTORSIZE * VECTORSIZE * sizeof(int), hipMemcpyHostToDevice);

     // Launch the kernel on the GPU
    int threadsPerBlock = 256;
    int blocksPerGrid = (VECTORSIZE + threadsPerBlock - 1) / threadsPerBlock;
    gettimeofday(&start_time, NULL);
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, VECTORSIZE);
    gettimeofday(&stop_time, NULL);

     // Copy the result from device to host
    hipMemcpy(C, d_C, VECTORSIZE * VECTORSIZE * sizeof(int), hipMemcpyDeviceToHost);
     exe_time = (stop_time.tv_sec + (stop_time.tv_usec / 1000000.0)) - (start_time.tv_sec + (start_time.tv_usec / 1000000.0));

     //print the data
  /*  printf("\n C matrix:\n");
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            printf("\t%d ", C[i * VECTORSIZE + j]);
        }
        printf("\n");
    }*/
    printf("\n Execution time is = %lf seconds\n", exe_time);
     printf("\nProgram exit!\n");

     //Free arrays
    free(A);
    free(B);
    free(C);

     // Free memory on the device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
     return 0;
}
