
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define VECTORSIZE 10000

__global__ void matrix_multiply(int *A, int *B, int *C, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j, k, sum;
    if (i < size)
    {
        for (j = 0; j < size; j++)
        {
            sum = 0;
            for (k = 0; k < size; k++)
            {
                sum += A[i * size + k] * B[k * size + j];
            }
            C[i * size + j] = sum;
        }
    }
}
int main(int argc, char **argv)
{
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    double exe_time;
    struct timeval stop_time, start_time;
     //Allocate and initialize the arrays
    A = (int *)malloc(VECTORSIZE * VECTORSIZE * sizeof(int));
    B = (int *)malloc(VECTORSIZE * VECTORSIZE * sizeof(int));
    C = (int *)malloc(VECTORSIZE * VECTORSIZE * sizeof(int));
     //Initialize data to some value
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            A[i * VECTORSIZE + j] = 1;
            B[i * VECTORSIZE + j] = 2;
            C[i * VECTORSIZE + j] = 0;
        }
    }
     //print the data
  /*  printf("\nInitial data: \n");
    printf("\n A matrix:\n");
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            printf("\t%d ", A[i * VECTORSIZE + j]);
        }
        printf("\n");
    }
    printf("\n B matrix:\n");
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            printf("\t%d ", B[i * VECTORSIZE + j]);
        }
        printf("\n");
    }*/
     //Allocate memory on the device
    hipMalloc((void **)&d_A, VECTORSIZE * VECTORSIZE * sizeof(int));
    hipMalloc((void **)&d_B, VECTORSIZE * VECTORSIZE * sizeof(int));
    hipMalloc((void **)&d_C, VECTORSIZE * VECTORSIZE * sizeof(int));

     // Copy data from host to device
    hipMemcpy(d_A, A, VECTORSIZE * VECTORSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, VECTORSIZE * VECTORSIZE * sizeof(int), hipMemcpyHostToDevice);

     // Launch the kernel on the GPU
    int threadsPerBlock = 256;
    int blocksPerGrid = (VECTORSIZE + threadsPerBlock - 1) / threadsPerBlock;
    gettimeofday(&start_time, NULL);
    matrix_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, VECTORSIZE);
    gettimeofday(&stop_time, NULL);

     // Copy the result from device to host
    hipMemcpy(C, d_C, VECTORSIZE * VECTORSIZE * sizeof(int), hipMemcpyDeviceToHost);
    exe_time = (stop_time.tv_sec + (stop_time.tv_usec / 1000000.0)) - (start_time.tv_sec + (start_time.tv_usec / 1000000.0));

     //print the data
  /*  printf("\n C matrix:\n");
    for (int i = 0; i < VECTORSIZE; i++)
    {
        for (int j = 0; j < VECTORSIZE; j++)
        {
            printf("\t%d ", C[i * VECTORSIZE + j]);
        }
        printf("\n");
    }*/
    printf("\nExecution time is = %lf seconds\n", exe_time);
    printf("\nProgram exit!\n");

     //Free arrays
    free(A);
    free(B);
    free(C);

     // Free memory on the device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

/*

This program performs matrix multiplication on a GPU using CUDA. It multiplies two matrices A and B of size 10,000 x 10,000 and stores the result in matrix C. The program also measures the execution time of the matrix multiplication operation.
Here's a step-by-step explanation of the code:
1. It includes necessary libraries and defines the size of the vectors to be used.
2. The  `matrix_multiply`  function is a CUDA kernel that performs the matrix multiplication. It calculates the index  `i`  for each thread and then checks if  `i`  is less than  `size`  (the size of the matrix). If it is, it performs the multiplication and sum operations for the elements of the matrices A and B, and stores the result in matrix C.
3. In the  `main`  function, it first allocates memory for the matrices A, B, and C on the host (CPU) using  `malloc` .
4. It initializes the elements of matrices A and B to 1 and 2 respectively, and the elements of matrix C to 0.
5. It allocates memory for matrices A, B, and C on the device (GPU) using  `cudaMalloc` .
6. It copies the data from the host to the device using  `cudaMemcpy` .
7. It calculates the number of blocks needed for the GPU execution and records the start time.
8. It launches the  `matrix_multiply`  kernel on the GPU.
9. It records the stop time after the kernel execution.
10. It copies the result (matrix C) from the device to the host using  `cudaMemcpy` .
11. It calculates the execution time and prints it out.
12. Finally, it frees the memory allocated for the matrices on the host and the device using  `free`  and  `cudaFree`  respectively.

*/