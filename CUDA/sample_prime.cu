#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
#include<hip/hip_runtime.h>
 #define N 1000000
 __global__ void is_prime(int* d_count)
{
    int flag = 0;
    for(int i=3; i<N; i++)
    {
        flag = 0;
        for(int j=2; j<i && flag == 0; j++)
        {
            if((i % j) == 0)
            {
                flag = 1;
            }
        }
        if(flag == 0)
        {
            atomicAdd(d_count, 1);
        }
    }
}
 int main()
{
    int count = 1;
    int* d_count;
    hipMalloc(&d_count, sizeof(int));
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
     clock_t start_time = clock();
    is_prime<<<1, 1>>>(d_count);
    hipDeviceSynchronize();
    clock_t stop_time = clock();
     hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_count);
     double exe_time = ((double) (stop_time - start_time)) / CLOCKS_PER_SEC;
    printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
     return 0;
}