
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define ARRSIZE 100000

__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid = threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}


int test_result(int *m, int *n, int *p, int *md, int *nd, int *pd, int num_threads)
{
        int size = num_threads*sizeof(int);
        int i, flag;
        flag = 0;
	    hipMalloc(&md, size);
	    hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	    hipMalloc(&nd, size);
	    hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	    hipMalloc(&pd, size);

	    dim3   DimGrid(1, 1);     
	    dim3   DimBlock(num_threads, 1);   


	    arradd<<< DimGrid,DimBlock >>>(md,nd,pd);

	    hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	    
	    for(i=0;i<num_threads;i++)
	    {
	        if(p[i] != 2*i)
	        {
	            flag = 1;
	            break;
	        }
	    }
	    return flag;
}

int main()
{
	//int size = ARRSIZE * sizeof(int);
	int m[ARRSIZE], n[ARRSIZE], p[ARRSIZE],*md, *nd,*pd;
	int i=0;
    int start, end, num_threads;
    int flag;
	for(i=0; i<ARRSIZE; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}
    start = 1;
    end = 10000;
    printf("\n Choosing Start...");
    while(1)
    {   
        printf("\n Start = %d", start);
        flag = test_result(m, n, p, md, nd, pd, start);
	    if(flag != 0)   // Answers match
	    {
	        start = start/2;
	    }
	    else
	    {
	        break;
	    }
    }    
    
    printf("\n Choosing End...");
    while(1)
    {
	    printf("\n End = %d", end);
        flag = test_result(m, n, p, md, nd, pd, end);
	    if(flag == 0)   // Answers match
	    {
	        end = end *2;
	    }
	    else
	    {
	        break;
	    }	    
    }
    printf("\n Setting Start = %d and End = %d", start, end);    
    while(1)
    {
	    num_threads = (start + end)/2;
	    printf("\n Start = %d, End = %d, num_threads = %d", start, end,num_threads);
	    flag = test_result(m, n, p, md, nd, pd, num_threads);
	    if(flag == 0)   // Answers match
	    {
	        start = num_threads;
	        if((start == end) || ((end-start) == 1))
	        {
	            break;
	        }
	    }
	    else
	    {
	        end = num_threads;
	    }
    }
	printf("\nFinal value of num_threads = %d \n", num_threads);
		
	hipFree(md); 
	hipFree(nd);
	hipFree(pd);	
}

