
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<sys/time.h>
#define N 400

__global__ void calc_square(int* a, int* aa)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        aa[tid] = a[tid] * a[tid];
    }
}
 int main()
{
    int a[N], aa[N];
    int* d_a, *d_aa;
    double exe_time;
    struct timeval stop_time, start_time;
     // Initialize the input array
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_aa, N * sizeof(int));
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    gettimeofday(&start_time, NULL);
    calc_square <<< (N + 255) / 256, 256 >>> (d_a, d_aa);
    hipDeviceSynchronize();
    gettimeofday(&stop_time, NULL);	
    hipMemcpy(aa, d_aa, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_aa);

    for (int i = 0; i < N; i++)
    {
        printf("%d ", aa[i]);
    }
    printf("\n");

	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
    printf("Execution time: %lf seconds\n", exe_time);
    return 0;
}