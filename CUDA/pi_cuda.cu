
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define N 100000

__global__ void pi_calc(double *sum)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;	
	double x, step;
	if(myid<N)
	{
		step = 1.0/(double)N;
		x = (myid)*step;
		sum[myid] = 4.0/(1.0+x*x);
	}
}
/*
step = 1.0/(double)N;
		for(i=0; i<N; i++){
				x = (i)*step;
				sum = sum + 4.0/(1.0+x*x);
		}
		pi = step*sum;
*/
int main()
{
	double *sum, *sum_d;
	int i=0;
	double total = 0.0;
	double pi, step;
	double exe_time;
	step = 1.0/(double)N;
	struct timeval stop_time, start_time;
	
	sum = (double *)malloc(N*sizeof(double));
	hipMalloc(&sum_d, N*sizeof(double));
	
	gettimeofday(&start_time, NULL);
	
	int thds_per_block = 256;
	int num_blocks = (N/thds_per_block)+1;
	
	pi_calc<<< num_blocks,thds_per_block >>>(sum_d);
	hipMemcpy(sum, sum_d, N*sizeof(double), hipMemcpyDeviceToHost);
	
	for(i=0; i<N; i++)
	{
		total += sum[i];
	}
	pi = step*total;
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
		
	printf("\nValue of pi = %lf", pi);
	printf("\nExecution time is = %lf seconds\n", exe_time);
	hipFree(sum_d); 
	free(sum);
}

/*

This code calculates the value of pi using the Monte Carlo method in parallel using CUDA. 
1. The code includes necessary header files: stdio.h, stdlib.h, and sys/time.h.
2. The constant N is defined with a value of 9000000.
3. The kernel function "pi_calc" is defined. It takes a pointer to a double variable "sum" as an argument.
4. Inside the kernel function, each thread calculates a portion of the sum required to calculate pi.
5. The myid variable is calculated based on the thread and block indices.
6. If myid is less than N, the thread calculates the value of x and the corresponding sum and stores it in the sum array.
7. The main function starts.
8. Various variables are declared including sum, sum_d (device memory for sum), i, total, pi, step, and exe_time.
9. The sum array is allocated memory using malloc.
10. Device memory for sum is allocated using cudaMalloc.
11. The start time is recorded using gettimeofday.
12. The number of threads per block is set to 256 and the number of blocks is calculated based on N and the number of threads per block.
13. The pi_calc kernel function is launched with the specified number of blocks and threads per block.
14. The sum array is copied back from device memory to host memory using cudaMemcpy.
15. The total sum is calculated by iterating over the sum array.
16. The final value of pi is calculated by multiplying the step value with the total sum.
17. The stop time is recorded using gettimeofday.
18. The execution time is calculated by subtracting the start time from the stop time.
19. The calculated value of pi and the execution time are printed.
20. Device memory for sum is freed using cudaFree.
21. Host memory for sum is freed using free.

*/