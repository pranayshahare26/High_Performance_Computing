
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define N 9000000

__global__ void pi_calc(double *sum)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;	
	double x, step;
	if(myid<N)
	{
	    step = 1.0/(double)N;
        x = (myid)*step;
        sum[myid] = 4.0/(1.0+x*x);
    }
}
/*
step = 1.0/(double)N;
        for(i=0; i<N; i++){
                x = (i)*step;
                sum = sum + 4.0/(1.0+x*x);
        }
        pi = step*sum;
*/
int main()
{
	double *sum, *sum_d;
	int i=0;
	double total = 0.0;
	double pi, step;
	double exe_time;
	step = 1.0/(double)N;
	struct timeval stop_time, start_time;
    
	sum = (double *)malloc(N*sizeof(double));
	hipMalloc(&sum_d, N*sizeof(double));
	
	
	gettimeofday(&start_time, NULL);
	
	int thds_per_block = 256;
	int num_blocks = (N/thds_per_block)+1;
	
	pi_calc<<< num_blocks,thds_per_block >>>(sum_d);
	hipMemcpy(sum, sum_d, N*sizeof(double), hipMemcpyDeviceToHost);
	
	for(i=0; i<N; i++)
	{
        total += sum[i];
    }
    pi = step*total;
    
    gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	    
    printf("\n pi = %lf and exe_time = %lf\n", pi, exe_time);	
    hipFree(sum_d); 
    free(sum);
}
