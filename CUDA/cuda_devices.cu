#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main() 
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
	{
	    printf("There is no device supporting CUDA\n");
	}
	int dev;
	for (dev = 0; dev < deviceCount; ++dev) 
	{
	    hipDeviceProp_t deviceProp;
	    hipGetDeviceProperties(&deviceProp, dev);
	    if (dev == 0) 
	    {
		    if (deviceProp.major < 1)
	        {	
				    printf("There is no device supporting CUDA.\n");
		    }
		    else if (deviceCount == 1)
		    {
		        printf("There is 1 device supporting CUDA\n");
		    }
		    else
	        {
				    printf("There are %d devices supporting CUDA\n", deviceCount);
		    }
	    }
	    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
	    printf("Major revision number:                         %d\n", deviceProp.major);
	    printf("Minor revision number:                         %d\n", deviceProp.minor);
	    printf("Total amount of global memory:                 %ld bytes\n", deviceProp.totalGlobalMem);
	    printf("Total amount of constant memory:               %ld bytes\n", deviceProp.totalConstMem); 
	    printf("Total amount of shared memory per block:       %ld bytes\n", deviceProp.sharedMemPerBlock);
	    printf("Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
	    printf("Warp size:                                     %d\n", deviceProp.warpSize);
		printf("Multiprocessor count:                          %d\n",deviceProp.multiProcessorCount );
	    printf("Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
	    printf("Maximum sizes of each dimension of a block:    %d x %d x %d\n", deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	    printf("Maximum sizes of each dimension of a grid:     %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],  deviceProp.maxGridSize[2]);
	    printf("Maximum memory pitch:                          %ld bytes\n", deviceProp.memPitch);
	    printf("Texture alignment:                             %ld bytes\n", deviceProp.textureAlignment);
	    printf("Clock rate:                                    %d kilohertz\n", deviceProp.clockRate);
	} 
}

/*

This code is used to retrieve information about the CUDA-enabled devices available on the system. 
It first calls the function  `hipGetDeviceCount`  to get the number of CUDA devices available.
If the number of devices is 0, it prints a message indicating that there are no devices supporting CUDA. 
Next, it enters a loop to iterate over each device. Inside the loop, it calls  `hipGetDeviceProperties`  to retrieve the properties of each device.
It then prints various information about each device, such as the device name, major and minor revision numbers, total global memory, total constant memory, shared memory per block, number of registers per block, warp size, multiprocessor count, maximum number of threads per block, maximum sizes of each dimension of a block and grid, maximum memory pitch, texture alignment, and clock rate.
The loop continues until all devices have been processed, and the program exits.

*/