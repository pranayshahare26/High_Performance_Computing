#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define N 1000000
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/
__global__ void is_prime(int* d_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= 3 && idx < N)
    {
        int flag = 0;
        for(int j=2; j<idx && flag == 0; j++)
        {
            if((idx % j) == 0)
            {
                flag = 1;
            }
        }
        if(flag == 0)
        {
            atomicAdd(d_count, 1);
        }
    }
}
int main()
{
    int count = 1;
    int* d_count;
    hipMalloc(&d_count, sizeof(int));
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    clock_t start_time = clock();
    is_prime<<<blocksPerGrid, threadsPerBlock>>>(d_count);
    
    hipDeviceSynchronize();
    clock_t stop_time = clock();
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_count);
    
    double exe_time = ((double) (stop_time - start_time)) / CLOCKS_PER_SEC;
    printf("\nNumber of prime numbers = %d \nExecution time is = %lf seconds\n", count, exe_time);
    return 0;
}

/*

The given code is a CUDA program that calculates the number of prime numbers from 3 to N, where N is defined as 1,000,000. It uses the concept of parallel computing to speed up the calculation process.
Here is a step-wise explanation of the code:
1. The code includes necessary header files and defines a constant N as 1,000,000.
2. The code defines a CUDA kernel function named "is_prime" that takes an integer pointer as an argument. This function will be executed on the GPU.
3. Inside the "is_prime" kernel function, each thread is assigned an index (idx) based on the block and thread dimensions. The index represents a number from 3 to N.
4. The kernel function checks if the index (idx) is divisible by any number from 2 to idx-1. If it is divisible, a flag is set to 1.
5. If the flag remains 0 after checking all the numbers, it means the index (idx) is a prime number. In that case, the atomicAdd function is used to increment the value pointed by d_count by 1.
6. The main function begins by initializing a count variable to 1. This count variable will store the number of prime numbers found.
7. Memory is allocated on the GPU for an integer variable d_count using hipMalloc.
8. The value of count is copied from the host to the device using hipMemcpy.
9. The block and thread dimensions are calculated based on the value of N. This determines the number of threads and blocks that will be used for parallel execution.
10. The start time is recorded using the clock function.
11. The "is_prime" kernel function is called with the calculated block and thread dimensions.
12. hipDeviceSynchronize is used to ensure that all the threads have finished executing before proceeding.
13. The stop time is recorded using the clock function.
14. The value of count is copied from the device to the host using hipMemcpy.
15. The memory allocated for d_count is freed using hipFree.
16. The execution time is calculated using the start and stop times.
17. Finally, the number of prime numbers and the execution time are printed on the console.
In summary, the code calculates the number of prime numbers from 3 to 1,000,000 using parallel computing with CUDA. It utilizes multiple threads on the GPU to perform the calculations in parallel, which reduces the execution time compared to a sequential approach.

*/