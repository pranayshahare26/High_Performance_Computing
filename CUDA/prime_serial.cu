#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>
#include<hip/hip_runtime.h>
#define N 1000000
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/
__global__ void is_prime(int* d_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= 3 && idx < N)
    {
        int flag = 0;
        for(int j=2; j<idx && flag == 0; j++)
        {
            if((idx % j) == 0)
            {
                flag = 1;
            }
        }
        if(flag == 0)
        {
            atomicAdd(d_count, 1);
        }
    }
}
int main()
{
    int count = 1;
    int* d_count;
    hipMalloc(&d_count, sizeof(int));
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    clock_t start_time = clock();
    is_prime<<<blocksPerGrid, threadsPerBlock>>>(d_count);
    
    hipDeviceSynchronize();
    clock_t stop_time = clock();
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_count);
    
    double exe_time = ((double) (stop_time - start_time)) / CLOCKS_PER_SEC;
    printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
    return 0;
}