
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define N 400

__global__ void calc_sum(double* a, double* b, double* c, double alpha)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + alpha * b[tid];
    }
}
int main()
{
    double a[N], b[N], c[N], alpha;
    double* d_a, *d_b, *d_c;
    alpha = 0.001;
    struct timeval stop_time, start_time;
    double exe_time;

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_c, N * sizeof(double));
    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    gettimeofday(&start_time, NULL);
    calc_sum<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, alpha);
    hipDeviceSynchronize();
    gettimeofday(&stop_time, NULL);

    hipMemcpy(c, d_c, N * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    for (int i = 0; i < N; i++)
    {
        printf("\t%lf", c[i]);
    }
    printf("\n");
    exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
    printf("Execution time: %lf seconds\n", exe_time);
    return 0;
}
